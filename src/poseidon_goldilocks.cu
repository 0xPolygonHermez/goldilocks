#include "hip/hip_runtime.h"
#include "gl64_t.cuh"
#include "cuda_utils.cuh"
#include "cuda_utils.hpp"
#include <omp.h>

#include "poseidon_goldilocks.hpp"
#include "merklehash_goldilocks.hpp"

#ifdef GPU_TIMING
#include "timer_gl.hpp"
#endif

typedef uint32_t u32;
typedef uint64_t u64;

// CUDA Threads per Block
#define TPB 64

#define MAX_WIDTH 12

/* --- Based on seq code --- */

__device__ __forceinline__ void pow7(gl64_t &x)
{
    gl64_t x2 = x * x;
    gl64_t x3 = x * x2;
    gl64_t x4 = x2 * x2;
    x = x3 * x4;
}

__device__ __forceinline__ void pow7_(gl64_t *x)
{
    gl64_t x2[SPONGE_WIDTH], x3[SPONGE_WIDTH], x4[SPONGE_WIDTH];
#pragma unroll
    for (int i = 0; i < SPONGE_WIDTH; ++i)
    {
        x2[i] = x[i] * x[i];
        x3[i] = x[i] * x2[i];
        x4[i] = x2[i] * x2[i];
        x[i] = x3[i] * x4[i];
    }
}

__device__ __forceinline__ void add_(gl64_t *x, const gl64_t C[SPONGE_WIDTH])
{
#pragma unroll
    for (int i = 0; i < SPONGE_WIDTH; ++i)
    {
        x[i] = x[i] + C[i];
    }
}

__device__ __forceinline__ void prod_(gl64_t *x, const gl64_t alpha, const gl64_t C[SPONGE_WIDTH])
{
#pragma unroll
    for (int i = 0; i < SPONGE_WIDTH; ++i)
    {
        x[i] = alpha * C[i];
    }
}

__device__ __forceinline__ void pow7add_(gl64_t *x, const gl64_t C[SPONGE_WIDTH])
{
    gl64_t x2[SPONGE_WIDTH], x3[SPONGE_WIDTH], x4[SPONGE_WIDTH];
#pragma unroll
    for (int i = 0; i < SPONGE_WIDTH; ++i)
    {
        x2[i] = x[i] * x[i];
        x3[i] = x[i] * x2[i];
        x4[i] = x2[i] * x2[i];
        x[i] = x3[i] * x4[i];
        x[i] = x[i] + C[i];
    }
}

__device__ __forceinline__ gl64_t dot_(gl64_t *x, const gl64_t C[SPONGE_WIDTH])
{
    gl64_t s0 = x[0] * C[0];
#pragma unroll
    for (int i = 1; i < SPONGE_WIDTH; i++)
    {
        s0 = s0 + x[i] * C[i];
    }
    return s0;
}

__device__ __forceinline__ void mvp_(gl64_t *state, const gl64_t* mat)
{
    gl64_t old_state[SPONGE_WIDTH];
    mymemcpy((uint64_t*)old_state, (uint64_t*)state, SPONGE_WIDTH);

    for (int i = 0; i < SPONGE_WIDTH; i++)
    {
        state[i] = mat[i] * old_state[0];
        for (int j = 1; j < SPONGE_WIDTH; j++)
        {
            state[i] = state[i] + (mat[12 * j + i] * old_state[j]);
        }
    }
}

// Constants defined in "poseidon_goldilocks_constants.hpp"
__device__ __constant__ uint64_t GPU_C[118];
__device__ __constant__ uint64_t GPU_S[507];
__device__ __constant__ uint64_t GPU_M[144];
__device__ __constant__ uint64_t GPU_P[144];

void init_gpu_const(int nDevices = 0)
{
    static int initialized = 0;

    if (initialized == 0)
    {
        initialized = 1;
        if (nDevices == 0)
        {
            CHECKCUDAERR(hipGetDeviceCount(&nDevices));
        }
        for (int i = 0; i < nDevices; i++)
        {
            CHECKCUDAERR(hipSetDevice(i));
            CHECKCUDAERR(hipMemcpyToSymbol(HIP_SYMBOL(GPU_M), PoseidonGoldilocksConstants::M, 144 * sizeof(uint64_t), 0, hipMemcpyHostToDevice));
            CHECKCUDAERR(hipMemcpyToSymbol(HIP_SYMBOL(GPU_P), PoseidonGoldilocksConstants::P, 144 * sizeof(uint64_t), 0, hipMemcpyHostToDevice));
            CHECKCUDAERR(hipMemcpyToSymbol(HIP_SYMBOL(GPU_C), PoseidonGoldilocksConstants::C, 118 * sizeof(uint64_t), 0, hipMemcpyHostToDevice));
            CHECKCUDAERR(hipMemcpyToSymbol(HIP_SYMBOL(GPU_S), PoseidonGoldilocksConstants::S, 507 * sizeof(uint64_t), 0, hipMemcpyHostToDevice));
        }
        CHECKCUDAERR(hipSetDevice(0));
    }
}

__device__ void hash_full_result_seq(gl64_t *state, const gl64_t *input, const gl64_t *GPU_C_GL, const gl64_t *GPU_S_GL, const gl64_t *GPU_M_GL, const gl64_t *GPU_P_GL)
{
    mymemcpy((uint64_t*)state, (uint64_t*)input, SPONGE_WIDTH);

    add_(state, GPU_C_GL);
    for (int r = 0; r < HALF_N_FULL_ROUNDS - 1; r++)
    {
        pow7add_(state, &(GPU_C_GL[(r + 1) * SPONGE_WIDTH]));
        mvp_(state, GPU_M_GL);
    }

    pow7add_(state, &(GPU_C_GL[(HALF_N_FULL_ROUNDS * SPONGE_WIDTH)]));
    mvp_(state, GPU_P_GL);

    for (int r = 0; r < N_PARTIAL_ROUNDS; r++)
    {
        pow7(state[0]);
        state[0] = state[0] + GPU_C_GL[(HALF_N_FULL_ROUNDS + 1) * SPONGE_WIDTH + r];
        gl64_t s0 = dot_(state, &(GPU_S_GL[(SPONGE_WIDTH * 2 - 1) * r]));
        gl64_t W_[SPONGE_WIDTH];
        prod_(W_, state[0], &(GPU_S_GL[(SPONGE_WIDTH * 2 - 1) * r + SPONGE_WIDTH - 1]));
        add_(state, W_);
        state[0] = s0;
    }

    for (int r = 0; r < HALF_N_FULL_ROUNDS - 1; r++)
    {
        pow7add_(state, &(GPU_C_GL[(HALF_N_FULL_ROUNDS + 1) * SPONGE_WIDTH + N_PARTIAL_ROUNDS + r * SPONGE_WIDTH]));
        mvp_(state, GPU_M_GL);
    }
    pow7_(&(state[0]));
    mvp_(state, GPU_M_GL);
}

/* --- integration --- */

__device__ void linear_hash_one(gl64_t *output, gl64_t *input, uint32_t size, int tid)
{
    u32 remaining = size;
    __shared__ gl64_t GPU_C_SM[118];
    __shared__ gl64_t GPU_S_SM[507];
    __shared__ gl64_t GPU_M_SM[144];
    __shared__ gl64_t GPU_P_SM[144];

    if (tid == 0)
    {
        mymemcpy((uint64_t*)GPU_C_SM, GPU_C, 118);
        mymemcpy((uint64_t*)GPU_S_SM, GPU_S, 507);
        mymemcpy((uint64_t*)GPU_M_SM, GPU_M, 144);
        mymemcpy((uint64_t*)GPU_P_SM, GPU_P, 144);
    }
    __syncthreads();

    gl64_t state[SPONGE_WIDTH];


    if (size <= CAPACITY)
    {
        mymemcpy((uint64_t*)output, (uint64_t*)input, size);
        mymemset((uint64_t*)&output[size], 0, (CAPACITY - size));
        return; // no need to hash
    }
    while (remaining)
    {
        if (remaining == size)
        {
            mymemset((uint64_t*)(state + RATE), 0, CAPACITY);
        }
        else
        {
            mymemcpy((uint64_t*)(state + RATE), (uint64_t*)state, CAPACITY);
        }

        u32 n = (remaining < RATE) ? remaining : RATE;
        mymemset((uint64_t*)&state[n], 0, (RATE - n));
        mymemcpy((uint64_t*)state, (uint64_t*)(input + (size - remaining)), n);
        hash_full_result_seq(state, state, GPU_C_SM, GPU_S_SM, GPU_M_SM, GPU_P_SM);
        remaining -= n;
    }
    mymemcpy((uint64_t*)output, (uint64_t*)state, CAPACITY);
}

__device__ void linear_partial_hash_one(gl64_t *input, uint32_t size, gl64_t *state, int tid)
{
    __shared__ gl64_t GPU_C_SM[118];
    __shared__ gl64_t GPU_S_SM[507];
    __shared__ gl64_t GPU_M_SM[144];
    __shared__ gl64_t GPU_P_SM[144];

    if (tid == 0)
    {
        mymemcpy((uint64_t*)GPU_C_SM, GPU_C, 118);
        mymemcpy((uint64_t*)GPU_S_SM, GPU_S, 507);
        mymemcpy((uint64_t*)GPU_M_SM, GPU_M, 144);
        mymemcpy((uint64_t*)GPU_P_SM, GPU_P, 144);
    }
    __syncthreads();

    u32 remaining = size;

    while (remaining)
    {
        mymemcpy((uint64_t*)(state + RATE), (uint64_t*)state, CAPACITY);
        u32 n = (remaining < RATE) ? remaining : RATE;
        mymemset((uint64_t*)&state[n], 0, (RATE - n));
        mymemcpy((uint64_t*)state, (uint64_t*)(input + (size - remaining)), n);
        hash_full_result_seq(state, state, GPU_C_SM, GPU_S_SM, GPU_M_SM, GPU_P_SM);
        remaining -= n;
    }
}

__global__ void linear_hash_gpu(uint64_t *output, uint64_t *input, uint32_t size, uint32_t num_rows)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_rows)
        return;

    gl64_t *inp = (gl64_t *)(input + tid * size);
    gl64_t *out = (gl64_t *)(output + tid * CAPACITY);
    linear_hash_one(out, inp, size, threadIdx.x);
}

__global__ void linear_partial_init_hash_gpu(uint64_t *gstate, int32_t num_rows)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_rows)
        return;

    gl64_t *state = (gl64_t *)(gstate + tid * SPONGE_WIDTH);
    memset(state, 0, SPONGE_WIDTH * sizeof(gl64_t));
}

__global__ void linear_partial_hash_gpu(uint64_t *input, uint32_t num_cols, uint32_t num_rows, uint64_t *gstate, uint32_t hash_per_thread = 1)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_rows)
        return;

    for (uint32_t i = 0; i < hash_per_thread; i++)
    {
        gl64_t *inp = (gl64_t *)(input + (tid * hash_per_thread + i) * num_cols);
        gl64_t *state = (gl64_t *)(gstate + (tid * hash_per_thread + i) * SPONGE_WIDTH);
        linear_partial_hash_one(inp, num_cols, state, threadIdx.x);
    }
}

__global__ void linear_partial_copy_hash_gpu(uint64_t *output, uint64_t *gstate, uint32_t num_cols, uint32_t num_rows)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_rows)
        return;

    gl64_t *state = (gl64_t *)(gstate + tid * SPONGE_WIDTH);
    gl64_t *out = (gl64_t *)(output + tid * CAPACITY);
    mymemcpy((uint64_t*)out, (uint64_t*)state, CAPACITY);
    /*
    if (num_cols > 0)
    {
        mymemcpy((uint64_t*)out, (uint64_t*)state, CAPACITY);
    }
    else
    {
        mymemset((uint64_t*)out, 0, CAPACITY);
    }
    */
}

__device__ void hash_one(gl64_t *state, gl64_t *const input, int tid)
{
    __shared__ gl64_t GPU_C_SM[118];
    __shared__ gl64_t GPU_S_SM[507];
    __shared__ gl64_t GPU_M_SM[144];
    __shared__ gl64_t GPU_P_SM[144];

    if (tid == 0)
    {
        mymemcpy((uint64_t*)GPU_C_SM, GPU_C, 118);
        mymemcpy((uint64_t*)GPU_S_SM, GPU_S, 507);
        mymemcpy((uint64_t*)GPU_M_SM, GPU_M, 144);
        mymemcpy((uint64_t*)GPU_P_SM, GPU_P, 144);
    }
    __syncthreads();

    gl64_t aux[SPONGE_WIDTH];
    hash_full_result_seq(aux, input, GPU_C_SM, GPU_S_SM, GPU_M_SM, GPU_P_SM);
    mymemcpy((uint64_t*)state, (uint64_t*)aux, CAPACITY);
}

__global__ void hash_gpu(uint32_t nextN, uint32_t nextIndex, uint32_t pending, uint64_t *cursor)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= nextN)
        return;

    gl64_t pol_input[SPONGE_WIDTH];
    mymemset((uint64_t*)pol_input, 0, SPONGE_WIDTH);
    mymemcpy((uint64_t*)pol_input, (uint64_t*)&cursor[nextIndex + tid * RATE], RATE);
    hash_one((gl64_t *)(&cursor[nextIndex + (pending + tid) * CAPACITY]), pol_input, threadIdx.x);
}

void merkletree_cuda_batch(Goldilocks::Element *tree, uint64_t *dst_gpu_tree, uint64_t *gpu_tree, Goldilocks::Element *input, uint64_t num_cols, uint64_t num_rows, uint64_t dim, uint32_t const gpu_id)
{
    hipStream_t gpu_stream;
    CHECKCUDAERR(hipSetDevice(gpu_id));
    CHECKCUDAERR(hipStreamCreate(&gpu_stream));
    hipDeviceProp_t prop;
    CHECKCUDAERR(hipGetDeviceProperties(&prop, gpu_id));
    size_t numElementsTree = num_rows * CAPACITY;
    size_t totalMemNeeded = num_rows * num_cols * dim * sizeof(uint64_t) + numElementsTree * sizeof(uint64_t);
    size_t maxMem = prop.totalGlobalMem * 8 / 10;
    size_t batches = (size_t)ceil(totalMemNeeded / (1.0 * maxMem));
    size_t rowsBatch = (size_t)ceil(num_rows / (1.0 * batches));
    size_t rowsLastBatch = num_rows % rowsBatch;
    if (rowsLastBatch > 0)
    {
        batches--;
    }

#ifdef FDEBUG
    printf("GPU max mem: %lu\n", prop.totalGlobalMem);
    printf("GPU max usable mem: %lu\n", maxMem);
    printf("Total needed mem: %lu\n", totalMemNeeded);
    printf("Batches %lu\n", batches);
    printf("Rows per batch %lu\n", rowsBatch);
    printf("Rows last batch %lu\n", rowsLastBatch);
#endif

    uint64_t *gpu_input;
    CHECKCUDAERR(hipMalloc(&gpu_input, rowsBatch * num_cols * dim * sizeof(uint64_t)));

    Goldilocks::Element *iptr = input;
    uint64_t *gtree_ptr = gpu_tree;
    for (uint32_t b = 0; b < batches; b++)
    {
        CHECKCUDAERR(hipMemcpyAsync(gpu_input, (uint64_t *)iptr, rowsBatch * num_cols * dim * sizeof(uint64_t), hipMemcpyHostToDevice, gpu_stream));
        iptr += (rowsBatch * num_cols * dim);
        linear_hash_gpu<<<ceil(rowsBatch / (1.0 * TPB)), TPB, 0, gpu_stream>>>(gtree_ptr, gpu_input, num_cols * dim, rowsBatch);
        gtree_ptr += (rowsBatch * CAPACITY);
    }
    if (rowsLastBatch > 0)
    {
        CHECKCUDAERR(hipMemcpyAsync(gpu_input, (uint64_t *)iptr, rowsLastBatch * num_cols * dim * sizeof(uint64_t), hipMemcpyHostToDevice, gpu_stream));
        linear_hash_gpu<<<ceil(rowsLastBatch / (1.0 * TPB)), TPB, 0, gpu_stream>>>(gtree_ptr, gpu_input, num_cols * dim, rowsLastBatch);
    }
    if (dst_gpu_tree != NULL)
    {
        CHECKCUDAERR(hipMemcpyPeerAsync(dst_gpu_tree, 0, gpu_tree, gpu_id, numElementsTree * sizeof(uint64_t), gpu_stream));
    }
    CHECKCUDAERR(hipStreamSynchronize(gpu_stream));
    CHECKCUDAERR(hipFree(gpu_input));
    CHECKCUDAERR(hipStreamDestroy(gpu_stream));
}

void merkletree_cuda_multi_gpu(Goldilocks::Element *tree, uint64_t *dst_gpu_tree, Goldilocks::Element *input, uint64_t num_cols, uint64_t num_rows, int nThreads, uint64_t dim, uint32_t const ngpu)
{
    uint64_t numElementsTree = MerklehashGoldilocks::getTreeNumElements(num_rows);
    // hipDeviceProp_t prop;
    // hipGetDeviceProperties(&prop, 0);
    // size_t totalMemNeeded = num_rows * num_cols * dim * sizeof(uint64_t) + numElementsTree * sizeof(uint64_t);
    // size_t maxMem = prop.totalGlobalMem * 8 / 10 * ngpu;
    // bool use_batch = (totalMemNeeded >= maxMem);
    bool use_batch = false;
    size_t rowsDevice = num_rows / ngpu;
    uint64_t numElementsTreeDevice = rowsDevice * CAPACITY;
    uint64_t **gpu_input = (uint64_t **)malloc(ngpu * sizeof(uint64_t *));
    uint64_t **gpu_tree = (uint64_t **)malloc(ngpu * sizeof(uint64_t *));
    hipStream_t *gpu_stream = (hipStream_t *)malloc(ngpu * sizeof(hipStream_t));
    assert(gpu_input != NULL);
    assert(gpu_tree != NULL);
    assert(gpu_stream != NULL);

#ifdef FDEBUG
    if (use_batch)
    {
        printf("Doing multi batch on multi gpu (%d GPUs)\n", ngpu);
    }
    else
    {
        printf("Doing multi gpu single batch (%d GPUs)\n", ngpu);
    }
    printf("Total rows: %lu\nRows per GPU: %lu\n", num_rows, rowsDevice);
#endif

    if (use_batch)
    {
#pragma omp parallel for num_threads(ngpu)
        for (uint32_t d = 0; d < ngpu; d++)
        {
            CHECKCUDAERR(hipSetDevice(d));
            CHECKCUDAERR(hipMalloc(&gpu_tree[d], numElementsTreeDevice * sizeof(uint64_t)));
            merkletree_cuda_batch(tree + (d * numElementsTreeDevice), dst_gpu_tree + (d * numElementsTreeDevice), gpu_tree[d], input + (d * rowsDevice * num_cols * dim), num_cols, rowsDevice, dim, d);
        }

#pragma omp parallel for num_threads(ngpu)
        for (uint32_t d = 0; d < ngpu; d++)
        {
            CHECKCUDAERR(hipSetDevice(d));
            CHECKCUDAERR(hipFree(gpu_tree[d]));
        }
    }
    else
    {
#ifdef GPU_TIMING
        TimerStart(merkletree_cuda_multi_gpu_copyToGPU);
#endif
#pragma omp parallel for num_threads(ngpu)
        for (uint32_t d = 0; d < ngpu; d++)
        {
            CHECKCUDAERR(hipSetDevice(d));
            CHECKCUDAERR(hipMalloc(&gpu_tree[d], numElementsTreeDevice * sizeof(uint64_t)));
            CHECKCUDAERR(hipMalloc(&gpu_input[d], rowsDevice * num_cols * dim * sizeof(uint64_t)));
            CHECKCUDAERR(hipStreamCreate(gpu_stream + d));
            CHECKCUDAERR(hipMemcpyAsync(gpu_input[d], (uint64_t *)(input + d * rowsDevice * num_cols * dim), rowsDevice * num_cols * dim * sizeof(uint64_t), hipMemcpyHostToDevice, gpu_stream[d]));
        }
#ifdef GPU_TIMING
        for (uint32_t d = 0; d < ngpu; d++)
        {
            CHECKCUDAERR(hipStreamSynchronize(gpu_stream[d]));
        }
        TimerStopAndLog(merkletree_cuda_multi_gpu_copyToGPU);
        TimerStart(merkletree_cuda_multi_gpu_kernel);
#endif
#pragma omp parallel for num_threads(ngpu)
        for (uint32_t d = 0; d < ngpu; d++)
        {
            linear_hash_gpu<<<ceil(rowsDevice / (1.0 * TPB)), TPB, 0, gpu_stream[d]>>>(gpu_tree[d], gpu_input[d], num_cols * dim, rowsDevice);
        }
#ifdef GPU_TIMING
        for (uint32_t d = 0; d < ngpu; d++)
        {
            CHECKCUDAERR(hipStreamSynchronize(gpu_stream[d]));
        }
        TimerStopAndLog(merkletree_cuda_multi_gpu_kernel);
        TimerStart(merkletree_cuda_multi_gpu_copyPeer2Peer);
#endif
#pragma omp parallel for num_threads(ngpu)
        for (uint32_t d = 0; d < ngpu; d++)
        {
            CHECKCUDAERR(hipMemcpyPeer(dst_gpu_tree + (d * numElementsTreeDevice), 0, gpu_tree[d], d, numElementsTreeDevice * sizeof(uint64_t)));
            CHECKCUDAERR(hipStreamSynchronize(gpu_stream[d]));
        }
#ifdef GPU_TIMING
        TimerStopAndLog(merkletree_cuda_multi_gpu_copyPeer2Peer);
        TimerStart(merkletree_cuda_multi_gpu_cleanup);
#endif
#pragma omp parallel for num_threads(ngpu)
        for (uint32_t d = 0; d < ngpu; d++)
        {
            CHECKCUDAERR(hipStreamSynchronize(gpu_stream[d]));
            CHECKCUDAERR(hipSetDevice(d));
            CHECKCUDAERR(hipStreamDestroy(gpu_stream[d]));
            CHECKCUDAERR(hipFree(gpu_input[d]));
            CHECKCUDAERR(hipFree(gpu_tree[d]));
        }
#ifdef GPU_TIMING
        TimerStopAndLog(merkletree_cuda_multi_gpu_cleanup);
#endif
    }

    free(gpu_input);
    free(gpu_tree);
    free(gpu_stream);
}

void PoseidonGoldilocks::merkletree_cuda_multi_gpu_full(Goldilocks::Element *tree, uint64_t** gpu_inputs, uint64_t** gpu_trees, void* v_gpu_streams, uint64_t num_cols, uint64_t num_rows, uint64_t num_rows_device, uint32_t const ngpu, uint64_t dim)
{
    hipStream_t* gpu_streams = (hipStream_t*)v_gpu_streams;
    uint64_t numElementsTree = MerklehashGoldilocks::getTreeNumElements(num_rows);
    uint64_t numElementsTreeDevice = num_rows_device * CAPACITY;

    uint64_t* gpu_final_tree;
    CHECKCUDAERR(hipSetDevice(0));
    CHECKCUDAERR(hipMalloc(&gpu_final_tree, numElementsTree * sizeof(uint64_t)));

    init_gpu_const(ngpu);

#ifdef GPU_TIMING
        TimerStart(merkletree_cuda_multi_gpu_kernel);
#endif
#pragma omp parallel for num_threads(ngpu)
        for (uint32_t d = 0; d < ngpu; d++)
        {
            CHECKCUDAERR(hipSetDevice(d));
            linear_hash_gpu<<<ceil(num_rows_device / (1.0 * TPB)), TPB, 0, gpu_streams[d]>>>(gpu_trees[d], gpu_inputs[d], num_cols * dim, num_rows_device);
        }
#ifdef GPU_TIMING
        for (uint32_t d = 0; d < ngpu; d++)
        {
            CHECKCUDAERR(hipStreamSynchronize(gpu_streams[d]));
        }
        TimerStopAndLog(merkletree_cuda_multi_gpu_kernel);
        TimerStart(merkletree_cuda_multi_gpu_copyPeer2Peer);
#endif
#pragma omp parallel for num_threads(ngpu)
        for (uint32_t d = 0; d < ngpu; d++)
        {
            CHECKCUDAERR(hipSetDevice(d));
            CHECKCUDAERR(hipMemcpyPeerAsync(gpu_final_tree + (d * numElementsTreeDevice), 0, gpu_trees[d], d, numElementsTreeDevice * sizeof(uint64_t), gpu_streams[d]));
            // CHECKCUDAERR(hipStreamSynchronize(gpu_streams[d]));
        }
#ifdef GPU_TIMING
        TimerStopAndLog(merkletree_cuda_multi_gpu_copyPeer2Peer);
        TimerStart(merkletree_cuda_multi_gpu_cleanup);
#endif
#pragma omp parallel for num_threads(ngpu)
        for (uint32_t d = 0; d < ngpu; d++)
        {
            CHECKCUDAERR(hipSetDevice(d));
            CHECKCUDAERR(hipStreamSynchronize(gpu_streams[d]));
        }
#ifdef GPU_TIMING
        TimerStopAndLog(merkletree_cuda_multi_gpu_cleanup);
#endif

    // Build the merkle tree
    CHECKCUDAERR(hipSetDevice(0));
    uint64_t pending = num_rows;
    uint64_t nextN = floor((pending - 1) / 2) + 1;
    uint64_t nextIndex = 0;
    int actual_tpb, actual_blks;
    while (pending > 1)
    {
        if (nextN < TPB)
        {
            actual_tpb = nextN;
            actual_blks = 1;
        }
        else
        {
            actual_tpb = TPB;
            actual_blks = nextN / TPB + 1;
        }
        hash_gpu<<<actual_blks, actual_tpb>>>(nextN, nextIndex, pending, gpu_final_tree);
        nextIndex += pending * CAPACITY;
        pending = pending / 2;
        nextN = floor((pending - 1) / 2) + 1;
    }
    uint64_t *buffer = get_pinned_mem();
    CHECKCUDAERR(hipMemcpy(buffer, gpu_final_tree, numElementsTree * sizeof(uint64_t), hipMemcpyDeviceToHost));
    uint64_t nthreads = omp_get_max_threads()/2;
    uint64_t piece = numElementsTree / nthreads;
    uint64_t last_piece = numElementsTree - (nthreads -1) * piece;
#pragma omp parallel for num_threads(nthreads)
    for (uint64_t d = 0; d < nthreads; d++) {
      uint64_t cur_piece = d == nthreads -1 ? last_piece: piece;
      memcpy(tree+d*piece, buffer+d*piece, cur_piece * sizeof(uint64_t));
    }

    CHECKCUDAERR(hipFree(gpu_final_tree));
}

void PoseidonGoldilocks::merkletree_cuda_multi_gpu_steps(uint64_t** gpu_inputs, uint64_t** gpu_trees, void* v_gpu_streams, uint64_t num_cols, uint64_t num_rows_device, uint32_t const ngpu, uint64_t dim)
{
    hipStream_t* gpu_streams = (hipStream_t*)v_gpu_streams;

    init_gpu_const(ngpu);

#ifdef GPU_TIMING
    TimerStart(merkletree_cuda_multi_gpu_kernel);
#endif
#pragma omp parallel for num_threads(ngpu)
    for (uint32_t d = 0; d < ngpu; d++)
    {
        CHECKCUDAERR(hipSetDevice(d));
        linear_hash_gpu<<<ceil(num_rows_device / (1.0 * TPB)), TPB, 0, gpu_streams[d]>>>(gpu_trees[d], gpu_inputs[d], num_cols * dim, num_rows_device);
    }
#ifdef GPU_TIMING
    for (uint32_t d = 0; d < ngpu; d++)
        {
            CHECKCUDAERR(hipStreamSynchronize(gpu_streams[d]));
        }
        TimerStopAndLog(merkletree_cuda_multi_gpu_kernel);
#endif
}

void PoseidonGoldilocks::merkletree_cuda_multi_gpu_final(Goldilocks::Element *tree, uint64_t* final_tree, void* v_gpu_streams, uint64_t num_rows)
{
    hipStream_t* gpu_streams = (hipStream_t*)v_gpu_streams;
    uint64_t numElementsTree = MerklehashGoldilocks::getTreeNumElements(num_rows);

    uint64_t* gpu_final_tree;
    CHECKCUDAERR(hipSetDevice(0));
    CHECKCUDAERR(hipMalloc(&gpu_final_tree, numElementsTree * sizeof(uint64_t)));

    init_gpu_const(0);

#ifdef GPU_TIMING
    TimerStart(merkletree_cuda_multi_gpu_final_copy2gpu);
#endif

    CHECKCUDAERR(hipMemcpyAsync(gpu_final_tree, final_tree, num_rows * CAPACITY * sizeof(uint64_t), hipMemcpyHostToDevice, gpu_streams[0]));

#ifdef GPU_TIMING
    TimerStopAndLog(merkletree_cuda_multi_gpu_final_copy2gpu);
#endif

    CHECKCUDAERR(hipStreamSynchronize(gpu_streams[0]));

    // Build the merkle tree
    CHECKCUDAERR(hipSetDevice(0));
    uint64_t pending = num_rows;
    uint64_t nextN = floor((pending - 1) / 2) + 1;
    uint64_t nextIndex = 0;
    int actual_tpb, actual_blks;
    while (pending > 1)
    {
        if (nextN < TPB)
        {
            actual_tpb = nextN;
            actual_blks = 1;
        }
        else
        {
            actual_tpb = TPB;
            actual_blks = nextN / TPB + 1;
        }
        hash_gpu<<<actual_blks, actual_tpb>>>(nextN, nextIndex, pending, gpu_final_tree);
        nextIndex += pending * CAPACITY;
        pending = pending / 2;
        nextN = floor((pending - 1) / 2) + 1;
    }
    uint64_t *buffer = get_pinned_mem();
    CHECKCUDAERR(hipMemcpy(buffer, gpu_final_tree, numElementsTree * sizeof(uint64_t), hipMemcpyDeviceToHost));
    uint64_t nthreads = omp_get_max_threads()/2;
    uint64_t piece = numElementsTree / nthreads;
    uint64_t last_piece = numElementsTree - (nthreads -1) * piece;
#pragma omp parallel for num_threads(nthreads)
    for (uint64_t d = 0; d < nthreads; d++) {
        uint64_t cur_piece = d == nthreads -1 ? last_piece: piece;
        memcpy(tree+d*piece, buffer+d*piece, cur_piece * sizeof(uint64_t));
    }

    CHECKCUDAERR(hipFree(gpu_final_tree));
}

void PoseidonGoldilocks::merkletree_cuda(Goldilocks::Element *tree, Goldilocks::Element *input, uint64_t num_cols, uint64_t num_rows, int nThreads, uint64_t dim)
{
    if (num_rows == 0)
    {
        return;
    }

    // is the input < 2 GB -> run on CPU
    if (num_rows * num_cols * dim <= (1ul << 32))
    {
#ifdef __AVX512__
        PoseidonGoldilocks::merkletree_avx512(tree, input, num_cols, num_rows, nThreads, dim);
#else
        PoseidonGoldilocks::merkletree_avx(tree, input, num_cols, num_rows, nThreads, dim);
#endif
        return;
    }

    uint64_t *gpu_tree = NULL;
    uint64_t numElementsTree = MerklehashGoldilocks::getTreeNumElements(num_rows); // includes CAPACITY
    init_gpu_const();
    u32 actual_tpb = TPB;
    u32 actual_blks = num_rows / TPB + 1;

    // is the input > 1 GB?
    if (num_rows * num_cols * dim > 134217728)
    {
        int nDevices;
        hipGetDeviceCount(&nDevices);
        if (nDevices > 1)
        {
            CHECKCUDAERR(hipSetDevice(0));
            CHECKCUDAERR(hipMalloc(&gpu_tree, numElementsTree * sizeof(uint64_t)));
            merkletree_cuda_multi_gpu(tree, gpu_tree, input, num_cols, num_rows, nThreads, dim, nDevices);
        }
        else
        {
            CHECKCUDAERR(hipSetDevice(0));
            CHECKCUDAERR(hipMalloc(&gpu_tree, numElementsTree * sizeof(uint64_t)));
            merkletree_cuda_batch(tree, NULL, gpu_tree, input, num_cols, num_rows, dim, 0);
        }
    }
    else
    {
#ifdef FDEBUG
        printf("On GPU, 1 batch\n");
#endif
        CHECKCUDAERR(hipSetDevice(0));
        uint64_t *gpu_input;
        CHECKCUDAERR(hipMalloc(&gpu_tree, numElementsTree * sizeof(uint64_t)));
        CHECKCUDAERR(hipMalloc(&gpu_input, num_rows * num_cols * dim * sizeof(uint64_t)));
        CHECKCUDAERR(hipMemcpyAsync(gpu_input, (uint64_t *)input, num_rows * num_cols * dim * sizeof(uint64_t), hipMemcpyHostToDevice));
        if (num_rows < TPB)
        {
            actual_tpb = num_rows;
            actual_blks = 1;
        }
        linear_hash_gpu<<<actual_blks, actual_tpb>>>(gpu_tree, gpu_input, num_cols * dim, num_rows);
        CHECKCUDAERR(hipFree(gpu_input));
    }

    // Build the merkle tree
    uint64_t pending = num_rows;
    uint64_t nextN = floor((pending - 1) / 2) + 1;
    uint64_t nextIndex = 0;
    while (pending > 1)
    {
        if (nextN < TPB)
        {
            actual_tpb = nextN;
            actual_blks = 1;
        }
        else
        {
            actual_tpb = TPB;
            actual_blks = nextN / TPB + 1;
        }
        hash_gpu<<<actual_blks, actual_tpb>>>(nextN, nextIndex, pending, gpu_tree);
        nextIndex += pending * CAPACITY;
        pending = pending / 2;
        nextN = floor((pending - 1) / 2) + 1;
    }
    CHECKCUDAERR(hipMemcpy(tree, gpu_tree, numElementsTree * sizeof(uint64_t), hipMemcpyDeviceToHost));
    CHECKCUDAERR(hipFree(gpu_tree));
}

 void PoseidonGoldilocks::partial_hash_init_gpu(uint64_t **state, uint32_t num_rows, uint32_t ngpus)
 {
    init_gpu_const();
    int nDevices;
    CHECKCUDAERR(hipGetDeviceCount(&nDevices));
    nDevices = (ngpus < nDevices) ? ngpus : nDevices;
    for (int i = 0; i < nDevices; i++)
    {
        CHECKCUDAERR(hipSetDevice(i));
        linear_partial_init_hash_gpu<<<ceil(num_rows / (1.0 * TPB)), TPB>>>(state[i], num_rows);
    }
    CHECKCUDAERR(hipSetDevice(0));
 }

void PoseidonGoldilocks::merkletree_cuda_gpudata(Goldilocks::Element *tree, uint64_t *gpu_input, uint64_t num_cols, uint64_t num_rows, int nThreads, uint64_t dim)
{
    if (num_rows == 0)
    {
        return;
    }

    uint64_t *gpu_tree = NULL;
    uint64_t numElementsTree = MerklehashGoldilocks::getTreeNumElements(num_rows); // includes CAPACITY
    init_gpu_const();
    u32 actual_tpb = TPB;
    u32 actual_blks = num_rows / TPB + 1;

    CHECKCUDAERR(hipSetDevice(0));
    CHECKCUDAERR(hipMalloc(&gpu_tree, numElementsTree * sizeof(uint64_t)));
    if (num_rows < TPB)
    {
        actual_tpb = num_rows;
        actual_blks = 1;
    }
    linear_hash_gpu<<<actual_blks, actual_tpb>>>(gpu_tree, gpu_input, num_cols * dim, num_rows);

    // Build the merkle tree
    uint64_t pending = num_rows;
    uint64_t nextN = floor((pending - 1) / 2) + 1;
    uint64_t nextIndex = 0;
    while (pending > 1)
    {
        if (nextN < TPB)
        {
            actual_tpb = nextN;
            actual_blks = 1;
        }
        else
        {
            actual_tpb = TPB;
            actual_blks = nextN / TPB + 1;
        }
        hash_gpu<<<actual_blks, actual_tpb>>>(nextN, nextIndex, pending, gpu_tree);
        nextIndex += pending * CAPACITY;
        pending = pending / 2;
        nextN = floor((pending - 1) / 2) + 1;
    }
    CHECKCUDAERR(hipMemcpy(tree, gpu_tree, numElementsTree * sizeof(uint64_t), hipMemcpyDeviceToHost));
    CHECKCUDAERR(hipFree(gpu_tree));
}

void PoseidonGoldilocks::partial_hash_gpu(uint64_t *input, uint32_t num_cols, uint32_t num_rows, uint64_t *state)
{
    linear_partial_hash_gpu<<<ceil((num_rows/2048)/(1.0 * TPB)), TPB>>>(input, num_cols, num_rows, state, 2048);
}
