#include "cuda_utils.hpp"
#include "cuda_utils.cuh"

#define MAX_GPUS 16

uint64_t *global_buffer;

void alloc_pinned_mem(uint64_t n)
{
  CHECKCUDAERR(hipHostMalloc(&global_buffer, n * sizeof(uint64_t)));
}

uint64_t* get_pinned_mem() {
  return global_buffer;
}

void free_pinned_mem()
{
    hipHostFree(global_buffer);
}

void warmup_all_gpus()
{
    uint64_t *gpu_a[MAX_GPUS];
    uint64_t size = (1 << 20);

    int nDevices = 0;
    CHECKCUDAERR(hipGetDeviceCount(&nDevices));

#pragma omp parallel for num_threads(nDevices)
    for (uint32_t d = 0; d < nDevices; d++)
    {
        CHECKCUDAERR(hipSetDevice(d));
        CHECKCUDAERR(hipMalloc(&gpu_a[d], size * sizeof(uint64_t)));
    }
#pragma omp parallel for num_threads(nDevices)
    for (uint32_t d = 0; d < nDevices; d++)
    {
        CHECKCUDAERR(hipSetDevice(d));
        CHECKCUDAERR(hipFree(gpu_a[d]));
    }
}
